#include "hip/hip_runtime.h"

#include <streams/AM.h>
#include <common/Common_Deffinitions.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>

extern "C"
struct am_demod_point
{
    const RADIO_DATA_TYPE m_threshold; ///< Decision Threshold

    /// Constructor
    am_demod_point(RADIO_DATA_TYPE threshold)
    : m_threshold(threshold)
    {
    }

    __host__ __device__
        RADIO_DATA_TYPE operator()(const RADIO_DATA_TYPE& x, const RADIO_DATA_TYPE& y) const {
            return (sqrt((x*x) + (y*y)) >= m_threshold) ? 1.0 : 0.0;
        }
};

extern "C"
__global__ void downsample(RADIO_DATA_TYPE* d_vec_x, uint8_t* output, size_t N, size_t ds)
{
    int tid = blockIdx.x;
    if(tid < N && tid % ds == 0)
    {
        output[tid/ds] = d_vec_x[tid];
    }
}

extern "C"
size_t am_gpu_demodulation(RADIO_DATA_TYPE* real, RADIO_DATA_TYPE* imag, uint8_t* output, size_t number_of_points, size_t ds)
{
    size_t N = number_of_points;
    std::cout << "AM GPU Demodulation is working" << std::endl;

    // Push data to the GPU for processing
    thrust::device_vector<RADIO_DATA_TYPE> d_vec_x(real, real+number_of_points);
    thrust::device_vector<RADIO_DATA_TYPE> d_vec_y(imag, imag+number_of_points);
    thrust::device_vector<uint8_t> d_output_vec(number_of_points/ds);

    // AM Demodulate Data
    thrust::transform(d_vec_x.begin(), d_vec_x.end(), d_vec_y.begin(), d_vec_y.begin(), am_demod_point(AM<uint8_t>::THRESHOLD));
    hipDeviceSynchronize(); // block until kernel is finished

    RADIO_DATA_TYPE* raw_input = thrust::raw_pointer_cast(d_vec_y.data()); // data returns to us in y not x
    uint8_t* raw_output = thrust::raw_pointer_cast(d_output_vec.data());

    // Downsample data on the device
    downsample<<<N, 1>>>( raw_input, raw_output, N, ds );
    hipDeviceSynchronize(); // block until kernel is finished
    // transfer data back to host
    thrust::copy(d_output_vec.begin(), d_output_vec.end(), output);


    if ( hipSuccess != hipGetLastError() )
    {
        std::cout << "[CUDA][am_gpu_demodulation] Error!" << std::endl;
    }

    return number_of_points/ds;
}
